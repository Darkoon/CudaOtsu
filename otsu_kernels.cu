#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "PngImage.h"

// CUDA Runtime
#include <hip/hip_runtime.h>

__global__ void calculateHistogram(unsigned int* histogram, unsigned char* rawPixels, long chunkSize, long totalPixels)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	int startPosition = id * chunkSize;
	for (int i = startPosition; i < (startPosition + chunkSize); i++) {
		if (i < totalPixels) {
			int pixelValue = (int)rawPixels[i];
			atomicAdd(&histogram[pixelValue], 1);
		}
	}
}

__global__ void computeClassVariances(unsigned int* histogram, double allProbabilitySum, long int totalPixels, double* betweenClassVariance)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	double firstClassProbability = 0, secondClassProbability = 0;
	double firstClassMean = 0, secondClassMean = 0;
	double firstProbabilitySum = 0;

	for (int t = 0; t < id; t++) {
		firstClassProbability += histogram[t];
		firstProbabilitySum += t * firstClassProbability;
	}

	secondClassProbability = totalPixels - firstClassProbability;

	firstClassMean = (double)firstProbabilitySum / (double)firstClassProbability;
	secondClassMean = (double)(allProbabilitySum - firstProbabilitySum) / (double)secondClassProbability;

	betweenClassVariance[id] = firstClassProbability * secondClassProbability * pow((firstClassMean - secondClassMean), 2);
}

__global__ void binarize(unsigned char* rawPixels, long totalPixels, long chunkSize, unsigned char threshold)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	int startPosition = id * chunkSize;
	for (int i = startPosition; i < (startPosition + chunkSize); i++) {
		if (i < totalPixels) {
			if ((int)rawPixels[i] > (int)threshold) {
				rawPixels[i] = PngImage::COLOR_WHITE;
			}
			else {
				rawPixels[i] = PngImage::COLOR_BLACK;
			}
		}
	}
}

extern "C" unsigned int* cudaCalculateHistogram(unsigned char* rawPixels, long totalPixels) {
	int threadsPerBlock = 256;
	int numBlocks = 256;

	//TODO: check hipGetDeviceProperties function!
	 
	unsigned int* hostHistogram = new unsigned int[PngImage::MAX_PIXEL_VALUE];
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		hostHistogram[i] = 0;
	}

	unsigned int* deviceHistogram;
	hipMalloc((void **)&deviceHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceHistogram, hostHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	unsigned char* deviceRawPixels;
	hipMalloc((void **)&deviceRawPixels, sizeof(unsigned char) * totalPixels);
	hipMemcpy(deviceRawPixels, rawPixels, sizeof(unsigned char) * totalPixels, hipMemcpyHostToDevice);

	long chunkSize = ceil(totalPixels / (threadsPerBlock * numBlocks));

	calculateHistogram<<<numBlocks, threadsPerBlock>>>(deviceHistogram, deviceRawPixels, chunkSize, totalPixels);

	hipMemcpy(hostHistogram, deviceHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE, hipMemcpyDeviceToHost);

	hipFree(deviceHistogram);
	hipFree(deviceRawPixels);

	return hostHistogram;
}

extern "C" unsigned char cudaFindThreshold(unsigned int* histogram, long int totalPixels) {
	int threadsPerBlock = 256;
	int numBlocks = 256;

	double allProbabilitySum = 0;
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		allProbabilitySum += i * histogram[i];
	}

	double* hostBetweenClassVariances = new double[PngImage::MAX_PIXEL_VALUE];
	for (int i = 0; i < PngImage::MAX_PIXEL_VALUE; i++) {
		hostBetweenClassVariances[i] = 0;
	}

	unsigned int* deviceHistogram;
	hipMalloc((void **)&deviceHistogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceHistogram, histogram, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	double* deviceBetweenClassVariances;
	hipMalloc((void **)&deviceBetweenClassVariances, sizeof(double) * PngImage::MAX_PIXEL_VALUE);
	hipMemcpy(deviceBetweenClassVariances, hostBetweenClassVariances, sizeof(double) * PngImage::MAX_PIXEL_VALUE, hipMemcpyHostToDevice);

	computeClassVariances<<<numBlocks, threadsPerBlock>>>(deviceHistogram, allProbabilitySum, totalPixels, deviceBetweenClassVariances);

	hipMemcpy(hostBetweenClassVariances, deviceBetweenClassVariances, sizeof(unsigned int) * PngImage::MAX_PIXEL_VALUE, hipMemcpyDeviceToHost);

	hipFree(deviceHistogram);
	hipFree(deviceBetweenClassVariances);

	double maxVariance = 0;
	unsigned char currentBestThreshold = 0;
	for (int t = 0; t < PngImage::MAX_PIXEL_VALUE; t++) {
		if (hostBetweenClassVariances[t] > maxVariance) {
			currentBestThreshold = t;
			maxVariance = hostBetweenClassVariances[t];
		}
	}

	return currentBestThreshold;
}

extern "C" unsigned char* cudaBinarize(unsigned char* rawPixels, long totalPixels, unsigned char threshold) {
	int threadsPerBlock = 256;
	int numBlocks = 256;

	unsigned char* hostRawPixels = new unsigned char[totalPixels];

	unsigned char* deviceRawPixels;
	hipMalloc((void **)&deviceRawPixels, sizeof(unsigned char) * totalPixels);
	hipMemcpy(deviceRawPixels, rawPixels, totalPixels * sizeof(unsigned char), hipMemcpyHostToDevice);

	long chunkSize = ceil(totalPixels / (threadsPerBlock * numBlocks));

	binarize<<<numBlocks, threadsPerBlock>>>(deviceRawPixels, totalPixels, chunkSize, threshold);

	hipMemcpy(hostRawPixels, deviceRawPixels, sizeof(unsigned char) * totalPixels, hipMemcpyDeviceToHost);

	hipFree(deviceRawPixels);

	return hostRawPixels;
}